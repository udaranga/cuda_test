#include "hip/hip_runtime.h"
#define NRANK 2
#define BATCH 10

#include "hip/hip_runtime.h"
#include ""
#include <hipfft/hipfft.h>
#include <stdio.h> 
#include <iomanip> 
#include <iostream>
#include <vector>

using namespace std;


 

const size_t NX = 4;
 const size_t NY = 6;

int main()
	{ 
// Input array (static) - host side 
	float h_in_data_temp[NX][NY] ={ 
 		 {0.7943 ,   0.6020 ,   0.7482  ,  0.9133  ,  0.9961 , 0.9261},
 	     {0.3112 ,   0.2630 ,   0.4505  ,  0.1524  ,  0.0782 ,  0.1782},
         {0.5285 ,   0.6541 ,   0.0838  ,  0.8258  ,  0.4427,  0.3842},
         {0.1656 ,   0.6892 ,   0.2290  ,  0.5383  ,  0.1067,  0.1712}
  		};

	// --------------------------------
	// Input array (dynamic) - host side 
	// Allocated Memory
// 	float **a = new float*[NX];  
// 	for (int r = 0; r < NX; ++r)  // this can be also done on GPU
//   		a[r] = new float[NY]; 
	float *a = new float[NX*NY];  
  	
	// Set the values
	size_t h_ipitch;
	for (int r = 0; r < NX; ++r)  // this can be also done on GPU
		{  	 
		for (int c = 0; c < NY; ++c)
			{	a[NY*r + c] = h_in_data_temp[r][c];	}
		}
	// --------------------------------

	// Output array - host side
	float2 *h_out_data_temp = new float2[NX*(NY/2+1)] ;
	//float2 h_out_data_temp[NX][NY/2+1] ;
    
	// Input and Output array - device side	
	hipfftHandle plan;
	hipfftReal *d_in_data;      
	hipfftComplex * d_out_data;
	int n[NRANK] = {NX, NY};

  	hipMalloc((void**)&d_in_data, sizeof(hipfftReal)*NX*NY);    
    hipMemcpy(d_in_data, a, sizeof(hipfftReal)*NY*NX, hipMemcpyHostToDevice);
    	 

//  Copy input array from Host to Device
	//size_t ipitch;
	//hipError_t  cudaStat1 = 	hipMallocPitch((void**)&d_in_data,&ipitch,NY*sizeof(hipfftReal),NX);	
	//cout << hipGetErrorString(cudaStat1) << endl;
	//hipError_t  cudaStat2 = 	hipMemcpy2D(d_in_data,ipitch,a,sizeof(float),sizeof(float),NY*NX,hipMemcpyHostToDevice);  //<------THIS DOESN"T WORK (DYNAMIC ARRAY) 
	////  //hipError_t  cudaStat2 = 	hipMemcpy2D(d_in_data,ipitch,h_in_data_temp,sizeof(float)*NY,sizeof(float)*NY,NX,hipMemcpyHostToDevice);  //<------THIS WORKS (STATIC ARRAY)
	//cout << hipGetErrorString(cudaStat2) << endl;

	hipMalloc((void**)&d_out_data, sizeof(hipfftComplex)*NX*(NY/2 + 1));
  

    //Performe the fft
    //hipfftPlanMany(&plan, NRANK, n,NULL, 1, 0,NULL, 1, 0,HIPFFT_R2C,BATCH);
	hipfftPlan2d(&plan, NX, NY , HIPFFT_R2C);
    cufftSetCompatibilityMode(plan, CUFFT_COMPATIBILITY_NATIVE);
    hipfftExecR2C(plan, d_in_data, d_out_data);
    hipDeviceSynchronize();

	// Copy d_in_data back to host
	//hipError_t  cudaStat4 = hipMemcpy2D(h_out_data_temp,(NY)*sizeof(float),d_in_data,ipitch,NY*sizeof(hipfftReal),NX,hipMemcpyDeviceToHost); 
	//hipMemcpy(h_out_data_temp,d_in_data ,  sizeof(hipfftReal)*NY*NX, hipMemcpyDeviceToHost);  // --- working direct in - out copy
	hipMemcpy(h_out_data_temp,d_out_data,  sizeof(hipfftComplex)*NX*(NY/2 + 1), hipMemcpyDeviceToHost);
	//cout << hipGetErrorString(cudaStat4) << endl;


	// Print the results
	for (int i = 0; i < NX; i++)	
		{
		for (int j =0 ; j< NY/2+1; j++)			 
			 printf(" %f + %fi",h_out_data_temp[i*(NY/2+1) + j].x ,h_out_data_temp[i*(NY/2+1) + j].y);
		printf("\n");	 
		}
	hipFree(d_in_data);

	return 0;
	}

 