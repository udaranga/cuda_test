#include "hip/hip_runtime.h"
#define NRANK 2
#define BATCH 10

#include "hip/hip_runtime.h"
#include ""
#include <hipfft/hipfft.h>
#include <stdio.h> 
#include <iomanip> 
#include <iostream>
#include <vector>

using namespace std;

const size_t NX = 4;
const size_t NY = 6;

int main()
	{ 
	// Input array (static) - host side 
	float h_in_data_static[NX][NY] ={ 
		{0.7943 ,   0.6020 ,   0.7482  ,  0.9133  ,  0.9961 , 0.9261},
		{0.3112 ,   0.2630 ,   0.4505  ,  0.1524  ,  0.0782 ,  0.1782},
		{0.5285 ,   0.6541 ,   0.0838  ,  0.8258  ,  0.4427,  0.3842},
		{0.1656 ,   0.6892 ,   0.2290  ,  0.5383  ,  0.1067,  0.1712}
		};

	// --------------------------------
	// Input array (dynamic) - host side 
	float *h_in_data_dynamic = new float[NX*NY];  

	// Set the values
	size_t h_ipitch;
	for (int r = 0; r < NX; ++r)  // this can be also done on GPU
		{  	 
		for (int c = 0; c < NY; ++c)
			{	h_in_data_dynamic[NY*r + c] = h_in_data_static[r][c];	}
		}
	// --------------------------------

	// Output array - host side
	float2 *h_out_data_temp = new float2[NX*(NY/2+1)] ; 


	// Input and Output array - device side	
	hipfftHandle plan;
	hipfftReal *d_in_data;      
	hipfftComplex * d_out_data;
	int n[NRANK] = {NX, NY};

	//  Copy input array from Host to Device
	size_t ipitch;
	hipError_t  cudaStat1 = 	hipMallocPitch((void**)&d_in_data,&ipitch,NY*sizeof(hipfftReal),NX);	
	cout << hipGetErrorString(cudaStat1) << endl;
	hipError_t  cudaStat2 = 	hipMemcpy2D(d_in_data,ipitch,h_in_data_dynamic,NY*sizeof(float),NY*sizeof(float),NX,hipMemcpyHostToDevice);   
	cout << hipGetErrorString(cudaStat2) << endl;

	//  Allocate memory for output array - device side
	size_t opitch;
	hipError_t  cudaStat3 = 	hipMallocPitch((void**)&d_out_data,&opitch,(NY/2+1)*sizeof(hipfftComplex),NX);	
	cout << hipGetErrorString(cudaStat3) << endl;
	
	//  Performe the fft
	int rank = 2; // 2D fft     
	int istride = 1, ostride = 1; // Stride lengths
	int idist = 1, odist = 1;     // Distance between batches
	int inembed[] = {ipitch, NX}; // Input size with pitch
	int onembed[] = {opitch, NX}; // Output size with pitch
	int batch = 1;
	hipfftPlanMany(&plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_R2C, batch);
	//hipfftPlan2d(&plan, NX, NY , HIPFFT_R2C);
	cufftSetCompatibilityMode(plan, CUFFT_COMPATIBILITY_NATIVE);
	hipfftExecR2C(plan, d_in_data, d_out_data);
	hipDeviceSynchronize();

	// Copy d_in_data back from device to host
	hipError_t  cudaStat4 = hipMemcpy2D(h_out_data_temp,(NY/2+1)*sizeof(float2), d_out_data, opitch, (NY/2+1)*sizeof(hipfftComplex), NX, hipMemcpyDeviceToHost); 
	cout << hipGetErrorString(cudaStat4) << endl;
	
	// Print the results
	for (int i = 0; i < NX; i++)	
		{
		for (int j =0 ; j< NY/2 + 1; j++)		
			printf(" %f + %fi",h_out_data_temp[i*(NY/2+1) + j].x ,h_out_data_temp[i*(NY/2+1) + j].y);
		printf("\n");	 
		}
	hipFree(d_in_data);

	return 0;
	}

